
#include "simulation.h"

#include <iomanip>

CellID Simulation::getCellIdFromPosition(Vec2f position)
{
	int cellX = (position.x > 0.0f) ? ((position.x + (gridWidth * 0.5f)) / gridWidth) : ((position.x - (gridWidth * 0.5f)) / gridWidth);
	int cellY = (position.y > 0.0f) ? ((position.y + (gridWidth * 0.5f)) / gridWidth) : ((position.y - (gridWidth * 0.5f)) / gridWidth);
	return { cellX, cellY };
}

unsigned int Simulation::hashCell(CellID cellId)
{
	unsigned int a = (unsigned int)cellId.cellX * 15823;
	unsigned int b = (unsigned int)cellId.cellY * 9737333;
	return a + b;
}

unsigned int Simulation::keyFromHash(unsigned int hash)
{
	return hash % (unsigned int)numUniqueCellKeys;
}

void Simulation::buildSpatialPartition()
{
	ballKeysProfiler.start();
	{
		profiler.start(); host_keys.resize(numBalls); profiler.stop(); time_resizeHostKeys = profiler.time();
		profiler.start(); device_keys.resize(numBalls); profiler.stop(); time_resizeDeviceKeys = profiler.time();

		profiler.start();
		for (int i = 0; i < numBalls; i++)
		{
			BallCellKeyPair& ballCellKeyPair = ballKeyPairs[i];
			unsigned int ballIndex = ballCellKeyPair.ballIndex;
			unsigned int key = keyFromHash(hashCell(getCellIdFromPosition(balls[ballCellKeyPair.ballIndex].currPos)));
			key = (key << 16);
			host_keys[i] = (key | ballIndex);
		}
		profiler.stop(); time_constructingUnorderedBallKeyPairs = profiler.time();

		numMembersOfHost = host_keys.size();
		numMembersOfDevice = device_keys.size();
		profiler.start(); thrust::copy(host_keys.begin(), host_keys.end(), device_keys.begin()); profiler.stop(); time_memcpyHostToDevice = profiler.time();
		profiler.start(); thrust::sort(device_keys.begin(), device_keys.end()); profiler.stop(); time_GPU_SORT = profiler.time();
		hipDeviceSynchronize();
		profiler.start(); thrust::copy(device_keys.begin(), device_keys.end(), host_keys.begin()); profiler.stop(); time_memcpyDeviceToHost = profiler.time();

		profiler.start();
		for (int i = 0; i < numBalls; i++)
		{
			unsigned int index = host_keys[i] & 0xFFFF;
			unsigned int key = host_keys[i] >> 16;
			BallCellKeyPair& ballCellKey = ballKeyPairs[i];
			ballCellKey.ballIndex = index;
			ballCellKey.cellKey = key;
		}
		profiler.stop(); time_unpackingSortedBallKeyPairs = profiler.time();
	}
	ballKeysProfiler.stop();

	startIndicesProfiler.start();
	{
		for (int i = 0; i < numUniqueCellKeys; i++)
		{
			CellProperties cellProperty;
			startIndices[i] = cellProperty;
		}

		for (int i = 0; i < numBalls; i++)
		{
			BallCellKeyPair& curr = ballKeyPairs[i];
			CellProperties& cellProperty = startIndices[curr.cellKey];
			if (cellProperty.size == 0)
				cellProperty.startIndex = i;
			cellProperty.size++;
		}
	}
	startIndicesProfiler.stop();
}

